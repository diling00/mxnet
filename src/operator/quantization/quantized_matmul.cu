/*!
 * Copyright (c) 2017 by Contributors
 * \file quantized_matmul.cu
 * \brief
 * \author Ziheng Jiang
*/
#include "./quantized_matmul-inl.h"
#include "./quantization_utils.h"
#include "../mxnet_op.h"

namespace mxnet {
namespace op {


struct CastStruct {
  MSHADOW_XINLINE static void Map(int i, int32_t *out, const float *in) {
    out[i] = static_cast<int32_t>(in[i]);
  }
};

template<typename SrcType, typename DstType, typename CmpType>
class QuantizedMatmulCublasOp : public Operator {
 public:
  explicit QuantizedMatmulCublasOp(const Context& ctx,
                                   const std::vector<TShape>& in_shape,
                                   const std::vector<TShape>& out_shape,
                                   const QuantizedMatmulParam& param) {
    src_type_ = mshadow::DataType<SrcType>::kCudaFlag;
    dst_type_ = mshadow::DataType<DstType>::kCudaFlag;
    cmp_type_ = mshadow::DataType<CmpType>::kCudaFlag;
    alpha_ = 1.0f;
    beta_  = 0.0f;
  }

  ~QuantizedMatmulCublasOp() {
  }

  virtual void Forward(const OpContext &ctx,
                       const std::vector<TBlob> &in_data,
                       const std::vector<OpReqType> &req,
                       const std::vector<TBlob> &out_data,
                       const std::vector<TBlob> &aux_args) {
    using namespace mshadow;
    CHECK_EQ(in_data.size(), 6U);
    CHECK_EQ(out_data.size(), 3U);
    Stream<gpu> *s = ctx.get_stream<gpu>();
    CHECK_EQ(s->blas_handle_ownership_, Stream<gpu>::OwnHandle);
    const TBlob& data   =  in_data[0];
    const TBlob& weight =  in_data[1];
    const TBlob& out    = out_data[0];
    TShape dshape = data.shape_;
    TShape wshape = weight.shape_;
    TShape oshape = out.shape_;

    // row_C = col_C(T) = cublas(col_B(T) * col_A(T)) = cublas(row_B, row_A)
    int m = dshape[0], n = dshape[1], k = wshape[1];
    CUBLAS_CALL(hipblasGemmEx(s->blas_handle_,
                             HIPBLAS_OP_N,
                             HIPBLAS_OP_N,
                             k,
                             m,
                             n,
                             &alpha_,
                             weight.dptr_,
                             src_type_,
                             k,
                             data.dptr_,
                             src_type_,
                             n,
                             &beta_,
                             out.dptr_,
                             dst_type_,
                             k,
                             cmp_type_,
                             HIPBLAS_GEMM_DEFAULT));

    // temporary solution
    // TODO(ziheng) use GemmEx HIP_R_32I mode
    mxnet_op::Kernel<CastStruct, gpu>::Launch(s, out.Size(),
        out.dptr<int32_t>(), static_cast<float*>(out.dptr_));

    mxnet_op::Kernel<QuantizationRangeForMultiplicationStruct, gpu>::Launch(s, 1,
      out_data[1].dptr<float>(), out_data[2].dptr<float>(),
       in_data[2].dptr<float>(),  in_data[3].dptr<float>(),
       in_data[4].dptr<float>(),  in_data[5].dptr<float>());
  }

  virtual void Backward(const OpContext &ctx,
                        const std::vector<TBlob> &out_grad,
                        const std::vector<TBlob> &in_data,
                        const std::vector<TBlob> &out_data,
                        const std::vector<OpReqType> &req,
                        const std::vector<TBlob> &in_grad,
                        const std::vector<TBlob> &aux_args) {}


 private:
  float alpha_;
  float beta_;
  hipDataType src_type_;
  hipDataType dst_type_;
  hipDataType cmp_type_;

  hipDataType convertToCudaDataType(int dtype) {
    hipDataType converted = HIP_R_32F;
    MSHADOW_TYPE_SWITCH(dtype, mxDType, {
      converted = mshadow::DataType<mxDType>::kCudaFlag;
    })
    return converted;
  }
};  // class QuantizedMatmulCublasOp


template<>
Operator* CreateOp<gpu>(int dtype,
                        const Context& ctx,
                        const std::vector<TShape>& in_shape,
                        const std::vector<TShape>& out_shape,
                        const QuantizedMatmulParam& param) {
  Operator *op = NULL;
  op = new QuantizedMatmulCublasOp<int8_t, float, float>(ctx,
    in_shape, out_shape, param);
  return op;
}

}  // namespace op
}  // namespace mxnet

