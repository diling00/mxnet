/*!
 * Copyright (c) 2017 by Contributors
 * \file quantized_max_pool.cu
 * \brief
 * \author Ziheng Jiang
*/
#include <vector>
#include "./quantized_max_pool-inl.h"

namespace mxnet {
namespace op {

template<typename DType>
class QuantizedMaxPoolCuDNNOp : public Operator {
 public:
  explicit QuantizedMaxPoolCuDNNOp(QuantizedMaxPoolParam p) {
    param_ = p;
    init_cudnn_ = false;
    dtype_ = mshadow::DataType<DType>::kCudnnFlag;
    mode_ = HIPDNN_POOLING_MAX;
    nan_prop_ = HIPDNN_NOT_PROPAGATE_NAN;
  }

  ~QuantizedMaxPoolCuDNNOp() {
    if (init_cudnn_) {
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(in_desc_));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc_));
      CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pooling_desc_));
    }
  }

  virtual void Forward(const OpContext &ctx,
                       const std::vector<TBlob> &in_data,
                       const std::vector<OpReqType> &req,
                       const std::vector<TBlob> &out_data,
                       const std::vector<TBlob> &aux_args) {
    using namespace mshadow;
    CHECK_EQ(in_data.size(), 1U);
    CHECK_EQ(out_data.size(), 1U);
    Stream<gpu> *s = ctx.get_stream<gpu>();
    CHECK_EQ(s->dnn_handle_ownership_, mshadow::Stream<gpu>::OwnHandle);
    float alpha = 1.0f;
    float beta  = 0.0f;
    CHECK(param_.kernel.ndim() == 2) << "Only support 2D pooling";
    if (!init_cudnn_) this->Init(s, in_data, out_data);
    CUDNN_CALL(hipdnnPoolingForward(s->dnn_handle_,
                                   pooling_desc_,
                                   &alpha,
                                   in_desc_,
                                   in_data[0].dptr_,
                                   &beta,
                                   out_desc_,
                                   out_data[0].dptr_));
  }

  virtual void Backward(const OpContext &ctx,
                        const std::vector<TBlob> &out_grad,
                        const std::vector<TBlob> &in_data,
                        const std::vector<TBlob> &out_data,
                        const std::vector<OpReqType> &req,
                        const std::vector<TBlob> &in_grad,
                        const std::vector<TBlob> &aux_args) {
    LOG(FATAL) << "Only support pool forward";
  }

 private:
  inline void Init(mshadow::Stream<gpu> *s,
                   const std::vector<TBlob> &in_data,
                   const std::vector<TBlob> &out_data) {
    using namespace mshadow;
    CHECK(!init_cudnn_) << "Init should only be called when init_cudnn is false";
    CHECK_EQ(in_data.size(), 1U);
    CHECK_EQ(out_data.size(), 1U);
    CHECK(param_.kernel.ndim() == 2) << "only support 2d pooling";
    const TBlob& data = in_data[0];
    const TBlob& out  = out_data[0];
    TShape dshape = data.shape_;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pooling_desc_));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc_));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc_));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(in_desc_,
                                          HIPDNN_TENSOR_NCHW,
                                          dtype_,
                                          dshape[0],
                                          dshape[1],
                                          dshape[2],
                                          dshape[3]));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(out_desc_,
                                          HIPDNN_TENSOR_NCHW,
                                          dtype_,
                                          dshape[0],
                                          dshape[1],
                                          dshape[2],
                                          dshape[3]));
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(
      pooling_desc_,
      mode_,
      nan_prop_,
      param_.global_pool ? dshape[2] : param_.kernel[0],
      param_.global_pool ? dshape[3] : param_.kernel[1],
      param_.pad[0],
      param_.pad[1],
      param_.global_pool ? 1 : param_.stride[0],
      param_.global_pool ? 1 :param_.stride[1]));
  }
  bool init_cudnn_;
  hipdnnDataType_t dtype_;
  hipdnnHandle_t handle_;
  hipdnnPoolingMode_t mode_;
  hipdnnTensorDescriptor_t in_desc_;
  hipdnnTensorDescriptor_t out_desc_;
  hipdnnPoolingDescriptor_t pooling_desc_;
  hipdnnNanPropagation_t nan_prop_;
  QuantizedMaxPoolParam param_;
};  // class QuantizedMaxPoolCuDNNOp

template<>
Operator *CreateOp<gpu>(QuantizedMaxPoolParam param, int dtype) {
  Operator *op = NULL;
  MSHADOW_TYPE_SWITCH(dtype, DType, {
    op = new QuantizedMaxPoolCuDNNOp<DType>(param);
  });
  return op;
}

}  // namespace op
}  // namespace mxnet

