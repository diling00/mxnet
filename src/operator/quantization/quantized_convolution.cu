/*!
 * Copyright (c) 2017 by Contributors
 * \file quantized_convolution.cu
 * \brief
 * \author Ziheng Jiang
*/
#include "./quantized_convolution-inl.h"
#include "./quantization_utils.h"

namespace mxnet {
namespace op {

template<typename SrcType, typename DstType, typename CmpType>
class QuantizedConvolutionCuDNNOp : public Operator {
 public:
  explicit QuantizedConvolutionCuDNNOp(const Context& ctx,
                                       const std::vector<TShape>& in_shape,
                                       const std::vector<TShape>& out_shape,
                                       const QuantizedConvolutionParam& param) {
    param_ = param;
    src_type_ = mshadow::DataType<SrcType>::kCudnnFlag;
    cmp_type_ = mshadow::DataType<CmpType>::kCudnnFlag;
    algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
    format_ = HIPDNN_TENSOR_NHWC;
    init_temp_size_ = false;
    // 1024 MB
    workspace_limit_ = 1024;
    workspace_limit_ = (workspace_limit_ << 20) / sizeof(SrcType);
    InitDescriptors(ctx, in_shape, out_shape);
  }

  ~QuantizedConvolutionCuDNNOp() {
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc_));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(data_desc_));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc_));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc_));
  }

  virtual void Forward(const OpContext &ctx,
                       const std::vector<TBlob> &in_data,
                       const std::vector<OpReqType> &req,
                       const std::vector<TBlob> &out_data,
                       const std::vector<TBlob> &aux_args) {
    using namespace mshadow;
    CHECK_EQ(in_data.size(), 6U);
    CHECK_EQ(out_data.size(), 3U);
    Stream<gpu> *s = ctx.get_stream<gpu>();
    CHECK_EQ(s->dnn_handle_ownership_, Stream<gpu>::OwnHandle);

    TBlob data   = in_data[0];
    TBlob filter = in_data[1];
    TBlob out    = out_data[0];
    if (!init_temp_size_) GetTempSize(ctx);
    LOG(INFO) << "Resource Request: " << workspace_;
    Tensor<gpu, 1, SrcType> workspace =
      ctx.requested[0].get_space_typed<gpu, 1, SrcType>(mshadow::Shape1(workspace_), s);

    float alpha = 1.0f;
    float beta = 0.0f;
    LOG(INFO) << "CuDNN Forward";
    CUDNN_CALL(hipdnnConvolutionForward(s->dnn_handle_,
                                       &alpha,
                                       data_desc_,
                                       data.dptr_,
                                       filter_desc_,
                                       filter.dptr_,
                                       conv_desc_,
                                       algo_,
                                       workspace.dptr_,
                                       workspace_byte_,
                                       &beta,
                                       out_desc_,
                                       out.dptr_));

    mxnet_op::Kernel<quantization_range_for_multiplication, gpu>::Launch(s, 1,
      out_data[1].dptr<float>(), out_data[2].dptr<float>(),
       in_data[2].dptr<float>(),  in_data[3].dptr<float>(),
       in_data[4].dptr<float>(),  in_data[5].dptr<float>());
  }

  virtual void Backward(const OpContext &ctx,
                        const std::vector<TBlob> &out_grad,
                        const std::vector<TBlob> &in_data,
                        const std::vector<TBlob> &out_data,
                        const std::vector<OpReqType> &req,
                        const std::vector<TBlob> &in_grad,
                        const std::vector<TBlob> &aux_args) {}


  void InitDescriptors(const Context& ctx,
                       const std::vector<TShape>& in_shape,
                       const std::vector<TShape>& out_shape) {
    TShape dshape =  in_shape[0];
    TShape kshape =  in_shape[1];
    TShape oshape = out_shape[0];
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc_));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&data_desc_));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc_));
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc_));

    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_desc_,
                                               param_.pad[0],
                                               param_.pad[1],
                                               param_.stride[0],
                                               param_.stride[1],
                                               1,
                                               1,
                                               HIPDNN_CROSS_CORRELATION,
                                               cmp_type_));

    LOG(INFO) << "dshape: " << dshape
      << ", kshape: " << kshape
      << ", oshape: " << oshape;

    CUDNN_CALL(hipdnnSetTensor4dDescriptor(data_desc_,
                                          format_,
                                          src_type_,
                                          dshape[0],
                                          dshape[3],
                                          dshape[1],
                                          dshape[2]));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(out_desc_,
                                          format_,
                                          src_type_,
                                          oshape[0],
                                          oshape[3],
                                          oshape[1],
                                          oshape[2]));
    // input:  [NHWC](batch, in_height, in_width, in_channels)
    // filter: [HWNC](out_channels, filter_height, filter_width, in_channels)
    // output: [NHWC](batch, out_height, out_width, out_channels)
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_desc_,
                                          src_type_,
                                          format_,
                                          kshape[0],
                                          kshape[3],
                                          kshape[1],
                                          kshape[2]));
  }

  void GetTempSize(const OpContext& ctx) {
    CHECK(!init_temp_size_)
      << "GetTempSize should only be called once.";
    mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(s->dnn_handle_,
                                                       data_desc_,
                                                       filter_desc_,
                                                       conv_desc_,
                                                       out_desc_,
                                                       algo_,
                                                       &workspace_byte_));
    workspace_ = workspace_byte_ / sizeof(SrcType) + 1;
    init_temp_size_ = true;
    LOG(INFO) << "GetWorkspaceSize Done";
  }


 private:
  bool init_temp_size_ = false;
  size_t workspace_limit_;
  QuantizedConvolutionParam param_;
  size_t workspace_;
  size_t workspace_byte_;
  hipdnnDataType_t src_type_;
  hipdnnDataType_t cmp_type_;
  hipdnnTensorFormat_t format_;
  hipdnnConvolutionDescriptor_t conv_desc_;
  hipdnnTensorDescriptor_t data_desc_;
  hipdnnFilterDescriptor_t filter_desc_;
  hipdnnTensorDescriptor_t out_desc_;
  hipdnnConvolutionFwdAlgo_t algo_;

  hipdnnDataType_t convertToCuDNNDataType(int dtype) {
    hipdnnDataType_t converted = HIPDNN_DATA_FLOAT;
    // The following will always assign to `converted` or throw an exception.
    MSHADOW_TYPE_SWITCH(dtype, mxDType, {
      converted = mshadow::DataType<mxDType>::kCudnnFlag;
    })
    return converted;
  }

};  // class QuantizedReluCuDNNOp


template<>
Operator* CreateOp<gpu>(int dtype,
                        const Context& ctx,
                        const std::vector<TShape>& in_shape,
                        const std::vector<TShape>& out_shape,
                        const QuantizedConvolutionParam& param) {
  Operator *op = NULL;
  op = new QuantizedConvolutionCuDNNOp<int8_t, int8_t, int32_t>(ctx,
    in_shape, out_shape, param);
  return op;
}

}  // namespace op
}  // namespace mxnet

